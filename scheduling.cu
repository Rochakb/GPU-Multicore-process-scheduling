#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include "kernel.h"

int schedule(int N, int M, int* arrival_times, int* burst_times, int** cores_schedules, int* cs_lengths){
    int TA = 0;
    
    int arrival[20000], burst[20000], length[10000];             //copy arrival and burst
    for(int j = 0; j < N; j++){
      burst[j] = burst_times[j];
      arrival[j] = arrival_times[j];
    }
    
    for(int i=0; i<M; i++){                                     //malloc 2D array of core schedules
      cores_schedules[i] = (int*) malloc(20000 * sizeof(int*));
    }
   
    for(int i = 0; i < M; i++){                                  //initialization
      cs_lengths[i] = 0;                                         //cs_length zero                            
      length[i] = 0;                                            //cs_length zero  
    }
  
    for(int i=0;i<N;i++){
       
        int len[10000];
        int arr = arrival[i];
        for(int j=0;j<M;j++){
          len[j] = length[j];
          if(length[j]<arr)
            len[j] = 0;
        }
        int *minn = thrust::min_element(thrust::host,len,len+M);
        int min = minn - len;

        if(arrival[i] > length[min])                        //calculate cs_length
          length[min] = arrival[i] + burst[i] - 1;
        else
          length[min] = length[min] + burst[i];
        
        TA = TA + length[min] - arrival[i] + 1;             // calculate TA
        
        cores_schedules[min][cs_lengths[min]] = i;
        cs_lengths[min]++;
    }
    
    
    return TA;
}